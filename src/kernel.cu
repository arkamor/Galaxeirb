#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "functions.h"

struct position{
	float pox; //4o
	float poy; //4o
	float poz; //4o
};	

struct velocite{
	float vex; //4o
	float vey; //4o
	float vez; //4o
};

struct star{ //32o
	float mas; //4o
	struct position pos; //12o
	struct velocite vel; //12o
	int galax; //1 milk 0 andro //4o
};

__global__ void kernel_acc_calc(struct star *deviceIn, struct star *deviceOut){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	struct star p[Nb_de_pts];

	memcpy(p, deviceIn, sizeof(p));	
	//memcpy
	if (i < Nb_de_pts) { 
		float dist;
		for(int j=0;j<Nb_de_pts;j++){
			if(j != i){
				dist = sqrtf(_Square(p[j].pos.pox - p[i].pos.pox)+_Square(p[j].pos.poy - p[i].pos.poy)+_Square(p[j].pos.poz - p[i].pos.poz));
				if(dist < 1.0) dist = 1.0;
				p[i].vel.vex += ((p[j].pos.pox - p[i].pos.pox) * mass_factor_X_damping_factor * (1/(_Cube(dist))) * p[j].mas);
				p[i].vel.vey += ((p[j].pos.poy - p[i].pos.poy) * mass_factor_X_damping_factor * (1/(_Cube(dist))) * p[j].mas);
				p[i].vel.vez += ((p[j].pos.poz - p[i].pos.poz) * mass_factor_X_damping_factor * (1/(_Cube(dist))) * p[j].mas);
			}
		}
	}

	memcpy(deviceOut, p, sizeof(p));

}

void acc_calc(int nblocks, int nthreads, struct star * in_addr, struct star * out_addr){
	kernel_acc_calc<<<nblocks, nthreads>>>(in_addr,out_addr);
}



/*
__global__ void kernel_saxpy( int n, float a, float * x, float * y, float * z ) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n ) { 
		z[i] = a * x[i] + y [i];
	}
}

void saxpy(int nblocks, int nthreads, int n, float a, float * x, float * y, float * z){
	kernel_saxpy<<<nblocks, nthreads>>>(n, a, x, y, z);
}*/